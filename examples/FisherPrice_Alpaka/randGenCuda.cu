

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define NPART 200

__global__ void genRands()
{
    // iTh is the thread number we use this throughout 
    int iTh=threadIdx.x +  blockIdx.x * blockDim.x;  

    hiprandState local_rand_state;
    hiprand_init(1984, iTh, 0, &local_rand_state);
    hiprand_uniform(&local_rand_state);
}

int main() {

  int num_threads = NPART;
  int num_blocks = 1;

  genRands<<<num_blocks, num_threads>>>();
  hipDeviceSynchronize();


}



